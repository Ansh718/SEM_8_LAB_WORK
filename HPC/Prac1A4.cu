#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;
using namespace std::chrono;

// ==============================
// CUDA Kernel for Vector Addition
// ==============================
__global__ void vectorAdd(int *A, int *B, int *C, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size)
    {
        C[tid] = A[tid] + B[tid];
    }
}

// ==============================
// CUDA Kernel for Matrix Multiplication
// ==============================
__global__ void matrixMultiply(int *A, int *B, int *C, int M, int N, int K)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < K)
    {
        int sum = 0;
        for (int i = 0; i < N; ++i)
            sum += A[row * N + i] * B[i * K + col];
        C[row * K + col] = sum;
    }
}

// ==============================
// Sequential Vector Addition (CPU)
// ==============================
void sequentialVectorAdd(int *A, int *B, int *C, int size)
{
    for (int i = 0; i < size; ++i)
        C[i] = A[i] + B[i];
}

// ==============================
// Sequential Matrix Multiplication (CPU)
// ==============================
void sequentialMatrixMultiply(int *A, int *B, int *C, int M, int N, int K)
{
    for (int i = 0; i < M; ++i)
        for (int j = 0; j < K; ++j)
        {
            int sum = 0;
            for (int k = 0; k < N; ++k)
                sum += A[i * N + k] * B[k * K + j];
            C[i * K + j] = sum;
        }
}

// ==============================
// Main Function
// ==============================
int main()
{
    // ========== Vector Addition ==========
    int vecSize = 1 << 20; // 1 million elements

    int *h_A = new int[vecSize];
    int *h_B = new int[vecSize];
    int *h_C_seq = new int[vecSize];
    int *h_C_par = new int[vecSize];

    for (int i = 0; i < vecSize; ++i)
    {
        h_A[i] = i + 1;
        h_B[i] = i + 2;
    }

    int *d_A, *d_B, *d_C;
    size_t vecBytes = vecSize * sizeof(int);
    hipMalloc(&d_A, vecBytes);
    hipMalloc(&d_B, vecBytes);
    hipMalloc(&d_C, vecBytes);

    // Sequential vector addition
    auto start = high_resolution_clock::now();
    sequentialVectorAdd(h_A, h_B, h_C_seq, vecSize);
    auto end = high_resolution_clock::now();
    auto seqVecTime = duration_cast<milliseconds>(end - start).count();

    // Parallel vector addition (GPU)
    hipMemcpy(d_A, h_A, vecBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, vecBytes, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (vecSize + threads - 1) / threads;

    start = high_resolution_clock::now();
    vectorAdd<<<blocks, threads>>>(d_A, d_B, d_C, vecSize);
    hipMemcpy(h_C_par, d_C, vecBytes, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    end = high_resolution_clock::now();
    auto parVecTime = duration_cast<milliseconds>(end - start).count();

    cout << "\n===== Vector Addition Results =====\n";
    cout << "Sequential Time: " << seqVecTime << " ms\n";
    cout << "Parallel Time (CUDA): " << parVecTime << " ms\n";

    // ========== Matrix Multiplication ==========
    int M = 256, N = 256, K = 256;
    int *matA = new int[M * N];
    int *matB = new int[N * K];
    int *matC_seq = new int[M * K];
    int *matC_par = new int[M * K];

    for (int i = 0; i < M * N; ++i)
        matA[i] = i % 100;
    for (int i = 0; i < N * K; ++i)
        matB[i] = (i * 2) % 100;

    int *d_matA, *d_matB, *d_matC;
    size_t matASize = M * N * sizeof(int);
    size_t matBSize = N * K * sizeof(int);
    size_t matCSize = M * K * sizeof(int);

    hipMalloc(&d_matA, matASize);
    hipMalloc(&d_matB, matBSize);
    hipMalloc(&d_matC, matCSize);

    hipMemcpy(d_matA, matA, matASize, hipMemcpyHostToDevice);
    hipMemcpy(d_matB, matB, matBSize, hipMemcpyHostToDevice);

    dim3 threads2D(16, 16);
    dim3 blocks2D((K + 15) / 16, (M + 15) / 16);

    // Sequential matrix multiplication
    start = high_resolution_clock::now();
    sequentialMatrixMultiply(matA, matB, matC_seq, M, N, K);
    end = high_resolution_clock::now();
    auto seqMatTime = duration_cast<milliseconds>(end - start).count();

    // Parallel matrix multiplication (GPU)
    start = high_resolution_clock::now();
    matrixMultiply<<<blocks2D, threads2D>>>(d_matA, d_matB, d_matC, M, N, K);
    hipDeviceSynchronize();
    hipMemcpy(matC_par, d_matC, matCSize, hipMemcpyDeviceToHost);
    end = high_resolution_clock::now();
    auto parMatTime = duration_cast<milliseconds>(end - start).count();

    cout << "\n===== Matrix Multiplication Results =====\n";
    cout << "Sequential Time: " << seqMatTime << " ms\n";
    cout << "Parallel Time (CUDA): " << parMatTime << " ms\n";
    cout << "Speedup: " << (float)seqMatTime / parMatTime << "x\n";

    // ========== Cleanup ==========
    delete[] h_A;
    delete[] h_B;
    delete[] h_C_seq;
    delete[] h_C_par;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] matA;
    delete[] matB;
    delete[] matC_seq;
    delete[] matC_par;
    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);

    return 0;
}